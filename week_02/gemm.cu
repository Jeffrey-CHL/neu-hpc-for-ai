// gemm.cu
// Week 02 Assignment: Implement GEMM in CUDA
// D = alpha * (A @ B) + beta * C

#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime.h>

#ifndef TILE
#define TILE 16
#endif

// ---------------- CPU reference ----------------
void gemm_cpu(int M, int N, int K,
              const float* A, const float* B, const float* C,
              float* D, float alpha, float beta) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            float acc = 0.f;
            for (int k = 0; k < K; ++k) {
                acc += A[i*K + k] * B[k*N + j];
            }
            D[i*N + j] = alpha * acc + beta * C[i*N + j];
        }
    }
}

// ---------------- Naive CUDA kernel ----------------
__global__ void gemm_naive(int M, int N, int K,
                           const float* A, const float* B, const float* C,
                           float* D, float alpha, float beta) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M && col < N) {
        float acc = 0.f;
        for (int k = 0; k < K; ++k) {
            acc += A[row*K + k] * B[k*N + col];
        }
        D[row*N + col] = alpha * acc + beta * C[row*N + col];
    }
}

// ---------------- Tiled CUDA kernel ----------------
__global__ void gemm_tiled(int M, int N, int K,
                           const float* A, const float* B, const float* C,
                           float* D, float alpha, float beta) {
    __shared__ float As[TILE][TILE];
    __shared__ float Bs[TILE][TILE];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float acc = 0.f;
    for (int t = 0; t < (K + TILE - 1) / TILE; ++t) {
        int a_col = t * TILE + threadIdx.x;
        int b_row = t * TILE + threadIdx.y;

        As[threadIdx.y][threadIdx.x] =
            (row < M && a_col < K) ? A[row*K + a_col] : 0.f;
        Bs[threadIdx.y][threadIdx.x] =
            (b_row < K && col < N) ? B[b_row*N + col] : 0.f;

        __syncthreads();

        for (int k = 0; k < TILE; ++k) {
            acc += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < M && col < N) {
        D[row*N + col] = alpha * acc + beta * C[row*N + col];
    }
}

// ---------------- Utilities ----------------
inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(1);
    }
}
#define gpuCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

void init_random(float* x, int n, unsigned seed=123) {
    srand(seed);
    for (int i = 0; i < n; ++i) {
        x[i] = ((rand() % 2001) - 1000) / 1000.0f;
    }
}

float l2_rel_error(const float* a, const float* b, int n) {
    double num = 0.0, den = 0.0;
    for (int i = 0; i < n; ++i) {
        double diff = (double)a[i] - (double)b[i];
        num += diff * diff;
        den += (double)b[i] * (double)b[i];
    }
    return (float) sqrt((num + 1e-12) / (den + 1e-12));
}

// ---------------- Main ----------------
int main(int argc, char** argv) {
    int M = (argc > 1) ? atoi(argv[1]) : 1024;
    int N = (argc > 2) ? atoi(argv[2]) : 1024;
    int K = (argc > 3) ? atoi(argv[3]) : 1024;
    int mode = (argc > 4) ? atoi(argv[4]) : 1;  // 0=naive, 1=tiled
    int tile = (argc > 5) ? atoi(argv[5]) : TILE;

    float alpha = 1.0f, beta = 1.0f;
    printf("GEMM: D = alpha*A@B + beta*C\n");
    printf("Dims: A(%d x %d), B(%d x %d), C/D(%d x %d)\n", M,K,K,N,M,N);
    printf("Kernel: %s, TILE=%d\n", mode==0?"naive":"tiled", tile);

    size_t szA = (size_t)M*K, szB = (size_t)K*N, szC = (size_t)M*N;
    float *hA = (float*)malloc(szA*sizeof(float));
    float *hB = (float*)malloc(szB*sizeof(float));
    float *hC = (float*)malloc(szC*sizeof(float));
    float *hD_ref = (float*)malloc(szC*sizeof(float));
    float *hD = (float*)malloc(szC*sizeof(float));

    init_random(hA, szA, 1);
    init_random(hB, szB, 2);
    init_random(hC, szC, 3);

    gemm_cpu(M,N,K,hA,hB,hC,hD_ref,alpha,beta);

    float *dA,*dB,*dC,*dD;
    gpuCheck(hipMalloc(&dA, szA*sizeof(float)));
    gpuCheck(hipMalloc(&dB, szB*sizeof(float)));
    gpuCheck(hipMalloc(&dC, szC*sizeof(float)));
    gpuCheck(hipMalloc(&dD, szC*sizeof(float)));

    gpuCheck(hipMemcpy(dA, hA, szA*sizeof(float), hipMemcpyHostToDevice));
    gpuCheck(hipMemcpy(dB, hB, szB*sizeof(float), hipMemcpyHostToDevice));
    gpuCheck(hipMemcpy(dC, hC, szC*sizeof(float), hipMemcpyHostToDevice));

    dim3 block(tile, tile);
    dim3 grid((N + block.x - 1)/block.x, (M + block.y - 1)/block.y);

    const int iters = 10;
    hipEvent_t start, stop;
    gpuCheck(hipEventCreate(&start));
    gpuCheck(hipEventCreate(&stop));

    gpuCheck(hipEventRecord(start));
    for (int it = 0; it < iters; ++it) {
        if (mode == 0)
            gemm_naive<<<grid, block>>>(M,N,K,dA,dB,dC,dD,alpha,beta);
        else
            gemm_tiled<<<grid, block>>>(M,N,K,dA,dB,dC,dD,alpha,beta);
    }
    gpuCheck(hipEventRecord(stop));
    gpuCheck(hipEventSynchronize(stop));
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    ms /= iters;

    gpuCheck(hipMemcpy(hD, dD, szC*sizeof(float), hipMemcpyDeviceToHost));

    float rel = l2_rel_error(hD, hD_ref, (int)szC);
    printf("Relative L2 error: %.3e\n", rel);
    double flops = 2.0 * (double)M * (double)N * (double)K;
    double gflops = flops / (ms/1000.0) / 1e9;
    printf("Time: %.3f ms, Throughput: %.2f GFLOP/s\n", ms, gflops);

    hipEventDestroy(start); hipEventDestroy(stop);
    hipFree(dA); hipFree(dB); hipFree(dC); hipFree(dD);
    free(hA); free(hB); free(hC); free(hD); free(hD_ref);
    return 0;
}